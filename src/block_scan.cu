#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "common.h"

#define LENGTH 10
#define BLOCK_SIZE 1024

typedef float num_t;

const size_t ARRAY_SIZE = (LENGTH * sizeof(num_t));

void reset(num_t *input, size_t length) {
  for (size_t i = 0; i < LENGTH; i++) {
    input[i] = 0;
  }
}

void host_bscan(num_t *input, num_t *result, size_t length) {
  result[0] = input[0];
  for (size_t i = 1; i < length; i++) {
    result[i] = result[i - 1] + input[i];
  }
}

__global__ 
void single_thread_bscan(num_t *input, num_t *result, size_t length) {
  int index = GLOBAL_INDEX;

  if (index > 0) {
    return;
  }

  result[0] = input[0];
  for (size_t i = 1; i < length; i++) {
    result[i] = result[i - 1] + input[i];
  }
}

__global__
void hsh_nsm_bscan(num_t *input, num_t *result, size_t length) {
  // TODO: This should not use shared memory, but in order to solve the read/write conflict
  // between blocks, we need to create an array 2x the size of `length` - how to solve this
  // easily without shared memory?

  int index = GLOBAL_INDEX;

  for (int stride = 1; stride <= length / 2; stride *= 2) {
    __syncthreads();

    bool should_add = index >= stride && index < length;

    if (!should_add) {
      continue;
    }

    input[index] = input[index] + input[index - stride];
  }

  result[index] = input[index];
}

void test_function(void (*func)(num_t*, num_t*, size_t), num_t *input, num_t *truth) {
  // Set up device arrays
  num_t *device_input = NULL;
  CUDA_ERROR(hipMalloc((void **) &device_input, ARRAY_SIZE), "Couldn't allocate device_input");
  CUDA_ERROR(
      hipMemcpy(device_input, input, ARRAY_SIZE, hipMemcpyHostToDevice),
      "Couldn't copy to device_input");
  num_t *device_result = NULL;
  CUDA_ERROR(hipMalloc((void **) &device_result, ARRAY_SIZE), "Couldn't allocate device_result");

  // Run the function to test
  int num_blocks = ceil(float(LENGTH) / float(BLOCK_SIZE));
  func<<<num_blocks, BLOCK_SIZE>>>(device_input, device_result, LENGTH);

  // Wait for the function to finish
  hipDeviceSynchronize();

  // Copy the results into host memory
  num_t *result = (num_t *)malloc(ARRAY_SIZE);
  CUDA_ERROR(
      hipMemcpy(result, device_result, ARRAY_SIZE, hipMemcpyDeviceToHost),
      "Couldn't copy to result");

  // Check it against the truth
  size_t num_incorrect = 0;
  for (size_t i = 0; i < LENGTH; i++) {
    if (truth[i] != result[i]) {
      fprintf(
          stderr,
          "Incorrect value at index %d: Expected %f, got %f\n",
          i,
          truth[i],
          result[i]);
      num_incorrect++;
    }
  }
  printf("Number of incorrect results: %ld\n", num_incorrect);
}

void run_block_scan(int argc, char **argv) {
  num_t *input = (num_t *)malloc(ARRAY_SIZE);
  num_t *truth = (num_t *)malloc(ARRAY_SIZE);
  for (size_t i = 0; i < LENGTH; i++) {
    input[i] = 1;
  }
  host_bscan(input, truth, LENGTH);

  test_function(single_thread_bscan, input, truth);
}

